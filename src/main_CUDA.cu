#include "hip/hip_runtime.h"
//
// Created by erick on 6/21/25.
//
/*
 * Based on CSC materials from:
 *
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "utils.cuh"
#include "main_CUDA.cuh"

#include "Point.h"
// #include "pngwriter.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 *
 * \returns An index in the unrolled 1D array.
 */
// int __host__ __device__ getIndex(const int i, const int j, const int width)
// {
//     return i*width + j;
// }

__global__ void evolve_kernel(simulationConf* conf, Point* points, const float aTimesDt)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i > 0 && i < conf->nx - 1)
    {
        int j = threadIdx.y + blockIdx.y*blockDim.y;
        if (j > 0 && j < conf->ny - 1)
        {
            const int index = getIndex(i, j, conf->ny);
            float uij = conf->Un[index];
            float uim1j = conf->Un[getIndex(i-1, j, conf->ny)];
            float uijm1 = conf->Un[getIndex(i, j-1, conf->ny)];
            float uip1j = conf->Un[getIndex(i+1, j, conf->ny)];
            float uijp1 = conf->Un[getIndex(i, j+1, conf->ny)];

            // Explicit scheme
            conf->Unp1[index] = uij + aTimesDt * ( (uim1j - 2.0*uij + uip1j)/conf->dx2 + (uijm1 - 2.0*uij + uijp1)/conf->dy2 );
            points[index].T = conf->Unp1[index];
        }
    }
}

void mainCUDA(simulationConf* conf, Point* points)
{
    float* Un = conf->Un;
    float* Unp1 = conf->Unp1;
    int nx = conf->nx, ny = conf->ny, numSteps = conf->numSteps,
    outputEvery = conf->outputEvery, numElements = conf->numElements;
    float a= conf->a, dt= conf->dt;
    const char* output_filename = conf->output_filename_GPU;

    float* d_Un;
    float* d_Unp1;
    Point* d_points;
    simulationConf* d_conf;

    hipMalloc((void**)&d_Un, numElements*sizeof(float));
    hipMalloc((void**)&d_Unp1, numElements*sizeof(float));
    hipMalloc((void**)&d_points, numElements*sizeof(Point));
    hipMalloc((void**)&d_conf, 1*sizeof(simulationConf));

    dim3 numBlocks(nx/BLOCK_SIZE_X + 1, ny/BLOCK_SIZE_Y + 1);
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    // Timing
    clock_t start = clock();


    // Main loop
    for (int n = 0; n <= numSteps; n++)
    {
        hipMemcpy(d_Un, Un, numElements*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Unp1, Unp1, numElements*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_points, points, numElements*sizeof(Point), hipMemcpyHostToDevice);
        conf->Un = d_Un;
        conf->Unp1 = d_Unp1;
        hipMemcpy(d_conf, conf, 1*sizeof(simulationConf), hipMemcpyHostToDevice);

        evolve_kernel<<<numBlocks, threadsPerBlock>>>(d_conf, d_points, a*dt);

        hipMemcpy(Un, d_Un, numElements*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Unp1, d_Unp1, numElements*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(points, d_points, numElements*sizeof(Point), hipMemcpyDeviceToHost);
        hipMemcpy(conf, d_conf, 1*sizeof(simulationConf), hipMemcpyDeviceToHost);
        conf->Un = Un;
        conf->Unp1 = Unp1;

        // Write the output if needed
        if (n % outputEvery == 0)
        {
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                exit(0);
            }
            //char filename[64];
            //sprintf(filename, "heat_%04d.png", n);
            //save_png(h_Un, nx, ny, filename, 'c');
            printArray(points, numElements, output_filename, n);
        }
        // Swapping the pointers for the next timestep
        std::swap(Un, Unp1);
    }

    // Timing
    clock_t finish = clock();
    printf("[CUDA] It took %f seconds\n", (double)(finish - start) / CLOCKS_PER_SEC);;

    hipFree(d_Un);
    hipFree(d_Unp1);
    hipFree(d_points);
    hipFree(d_conf);
}